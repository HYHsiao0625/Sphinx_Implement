#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../include/cuda_kernels.h"

__global__ void convolutionKernel(double *encImg, double *encConvW, double *encConvLayer, int filter_dim, int img_rows, int img_cols, int filter_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < img_rows && j < img_cols) {
        for (int k = 0; k < filter_size; k++) {
            for (int l = 0; l < filter_size; l++) {
                encConvLayer[i * img_cols + j] += encImg[(i + k) * img_cols + (j + l)] * encConvW[k * filter_size + l];
            }
        }
    }
}